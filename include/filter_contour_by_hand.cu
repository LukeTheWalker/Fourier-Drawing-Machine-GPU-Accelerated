#include "hip/hip_runtime.h"
#ifndef FILTER_CONTOUR_BY_HAND_WRAPPER_H
#define FILTER_CONTOUR_BY_HAND_WRAPPER_H

#define PRINT_FLAGS 0
#define PROFILING_HAND 0

#include <hip/hip_runtime.h>

#include <vector>
#include <unordered_set>

#include "opencv2/core.hpp"
#include "opencv2/imgproc.hpp"
#include "contour.hpp"

#include "utils.cuh"
#include "filter_contour.cu"
#include "streamCompaction.cu"

using namespace std;

struct check_array_membership {
    __device__ int4 operator()(int gi, int4 * dat_x_arr, int4 * dat_y_arr, int4 * arr_x, int4 * arr_y, int n_quart_array) {
        int4 res = {0, 0, 0, 0};
        int4 dat_x = dat_x_arr[gi];
        int4 dat_y = dat_y_arr[gi];
        for (int i = 0; i < n_quart_array; i++){
            res.x = res.x || (dat_x.x == arr_x[i].x && dat_y.x == arr_y[i].x) || (dat_x.x == arr_x[i].y && dat_y.x == arr_y[i].y) || (dat_x.x == arr_x[i].z && dat_y.x == arr_y[i].z) || (dat_x.x == arr_x[i].w && dat_y.x == arr_y[i].w);
            res.y = res.y || (dat_x.y == arr_x[i].x && dat_y.y == arr_y[i].x) || (dat_x.y == arr_x[i].y && dat_y.y == arr_y[i].y) || (dat_x.y == arr_x[i].z && dat_y.y == arr_y[i].z) || (dat_x.y == arr_x[i].w && dat_y.y == arr_y[i].w);
            res.z = res.z || (dat_x.z == arr_x[i].x && dat_y.z == arr_y[i].x) || (dat_x.z == arr_x[i].y && dat_y.z == arr_y[i].y) || (dat_x.z == arr_x[i].z && dat_y.z == arr_y[i].z) || (dat_x.z == arr_x[i].w && dat_y.z == arr_y[i].w);
            res.w = res.w || (dat_x.w == arr_x[i].x && dat_y.w == arr_y[i].x) || (dat_x.w == arr_x[i].y && dat_y.w == arr_y[i].y) || (dat_x.w == arr_x[i].z && dat_y.w == arr_y[i].z) || (dat_x.w == arr_x[i].w && dat_y.w == arr_y[i].w);
        }
        return {!res.x, !res.y, !res.z, !res.w};
    }
};

void load_contours_to_device (int * d_contours_x, int * d_contours_y, vector<vector<Point>> &contours, Sizes * sizes) {
    int *h_contours_x, *h_contours_y;
    hipError_t err;

    h_contours_x = (int *)malloc(sizes->contours_linear_size * sizeof(int));
    h_contours_y = (int *)malloc(sizes->contours_linear_size * sizeof(int));

    int idx = 0;
    for (int i = 0; i < sizes->number_of_contours; i++){
        for (int j = 0; j < contours[i].size(); j++){
            h_contours_x[idx] = contours[i][j].x;
            h_contours_y[idx] = contours[i][j].y;
            idx++;
        }
    }
    
    err = hipMemcpy(d_contours_x, h_contours_x, sizes->contours_linear_size * sizeof(int), hipMemcpyHostToDevice); cuda_err_check(err, __FILE__, __LINE__);
    err = hipMemcpy(d_contours_y, h_contours_y, sizes->contours_linear_size * sizeof(int), hipMemcpyHostToDevice); cuda_err_check(err, __FILE__, __LINE__);
    
    free(h_contours_x);
    free(h_contours_y);   
}

void filter_contour_by_hand_wrapper(int * d_contours_x_out, int * d_contours_y_out, int * h_contours_sizes_out, vector<vector<Point>> &contours, unordered_set<Point, HashFunction> &_excluded_points, Sizes * sizes, int ngroups = 1024, int lws = 256){
    int *d_excluded_points_x, *d_excluded_points_y, *d_contours_x, *d_contours_y;
    int *h_excluded_points_x, *h_excluded_points_y;
    int *d_flags;
    int excluded_points_size = _excluded_points.size();
    hipError_t err;

    // allocate memory on the host for all the points
    h_excluded_points_x = (int *)malloc(excluded_points_size * sizeof(int));
    h_excluded_points_y = (int *)malloc(excluded_points_size * sizeof(int));
    
    err = hipMalloc((void **)&d_contours_x, sizes->contours_linear_size * sizeof(int)); cuda_err_check(err, __FILE__, __LINE__);
    err = hipMalloc((void **)&d_contours_y, sizes->contours_linear_size * sizeof(int)); cuda_err_check(err, __FILE__, __LINE__);

    load_contours_to_device(d_contours_x, d_contours_y, contours, sizes);

    int idx = 0;
    for (auto it = _excluded_points.begin(); it != _excluded_points.end(); it++){
        h_excluded_points_x[idx] = it->x;
        h_excluded_points_y[idx] = it->y;
        idx++;
    }

    err = hipMalloc((void **)&d_excluded_points_x, excluded_points_size * sizeof(int)); cuda_err_check(err, __FILE__, __LINE__);
    err = hipMalloc((void **)&d_excluded_points_y, excluded_points_size * sizeof(int)); cuda_err_check(err, __FILE__, __LINE__);
    err = hipMalloc((void **)&d_flags, sizes->contours_linear_size * sizeof(int)); cuda_err_check(err, __FILE__, __LINE__);

    err = hipMemcpy(d_excluded_points_x, h_excluded_points_x, excluded_points_size * sizeof(int), hipMemcpyHostToDevice); cuda_err_check(err, __FILE__, __LINE__);
    err = hipMemcpy(d_excluded_points_y, h_excluded_points_y, excluded_points_size * sizeof(int), hipMemcpyHostToDevice); cuda_err_check(err, __FILE__, __LINE__);

    int nquarts_flags = round_div_up(sizes->contours_linear_size, 4);
    int nquarts_excluded_points = round_div_up(excluded_points_size, 4);

    #if PROFILING_HAND
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    #endif

    compute_flags<check_array_membership><<<round_div_up(nquarts_flags, 256), 256>>>(nquarts_flags, (int4*)d_flags, (int4*)d_contours_x, (int4*)d_contours_y, (int4*)d_excluded_points_x, (int4*)d_excluded_points_y, nquarts_excluded_points);
    
    #if PROFILING_HAND
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("compute_flags hand time: %f\n", milliseconds);
    printf("GE/s: %f\n", (float)sizes->contours_linear_size / milliseconds / 1e6);
    printf("GB/s: %f\n", ((float)sizes->contours_linear_size * sizeof(int) * 3 + (float)sizes->contours_linear_size * excluded_points_size * sizeof(int) * 4)/ milliseconds / 1e6);
    #endif

    err = hipGetLastError(); cuda_err_check(err, __FILE__, __LINE__);
    err = hipDeviceSynchronize(); cuda_err_check(err, __FILE__, __LINE__);

    #if PRINT_FLAGS
    printf("Flags computed: ");
    print_array_dev(d_flags, sizes->contours_linear_size);
    printf("\n");
    #endif

    filter_contour(d_contours_x, d_contours_y, h_contours_sizes_out, d_contours_x_out, d_contours_y_out, d_flags, sizes, ngroups, lws);

    free(h_excluded_points_x);
    free(h_excluded_points_y);

    hipFree(d_contours_x);
    hipFree(d_contours_y);
    hipFree(d_excluded_points_x);
    hipFree(d_excluded_points_y);
    hipFree(d_flags);

    #if PROFILING_HAND
    hipEventDestroy(start);
    hipEventDestroy(stop);
    #endif

    return;
}

int test() {
    vector<vector<Point>> contours;
    unordered_set<Point, HashFunction> excluded_points;

    ifstream in("debug.txt");
    int n;
    in >> n;
    vector<int> contour_sizes;

    for (int i = 0; i < n; i++){
        int m;
        in >> m;
        contour_sizes.push_back(m);
    }

    for (int i = 0; i < n; i++){
        vector<Point> contour;
        for (int j = 0; j < contour_sizes[i]; j++){
            int x, y;
            in >> x >> y;
            contour.push_back(Point(x, y));
        }
        contours.push_back(contour);
    }

    int m;
    in >> m;
    for (int i = 0; i < m; i++){
        int x, y;
        in >> x >> y;
        excluded_points.insert(Point(x, y));
    }

    cerr << "MISSING IMPORTANT DATA" << endl;

    // filter_contour_by_hand_wrapper(contours, excluded_points, 256, 256);

    // for (int i = 0; i < contours.size(); i++){
    //     for (int j = 0; j < contours[i].size(); j++){
    //         printf("(%d, %d) ", contours[i][j].x, contours[i][j].y);
    //     }
    //     printf("\n");
    // }
    return 0;
}
#endif