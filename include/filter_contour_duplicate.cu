#include "hip/hip_runtime.h"
#ifndef FILTER_CONTOUR_DUPLICATE_H
#define FILTER_CONTOUR_DUPLICATE_H

#define PRINT_DUP_FLAGS 0
#define PROFILE_DUP 1
#define KERNEL_SIZE_DUP 64

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>
#include <unordered_set>

#include "opencv2/core.hpp"
#include "opencv2/imgproc.hpp"
#include "contour.hpp"

#include "utils.cuh"
#include "filter_contour.cu"
#include "streamCompaction.cu"
#include "merge_contours.cu"

__global__ void compute_duplicates_flags (point * d_contours, int * d_flags, uint64_t nquarts_linear_size){
    uint64_t gi = threadIdx.x + blockIdx.x * blockDim.x;
    uint64_t n_comparison = ((uint64_t)nquarts_linear_size * ((uint64_t)nquarts_linear_size - 1)) / 2;

    uint64_t quarts_1 = (uint64_t)nquarts_linear_size - 2 - floor(sqrt((double)-8*gi + 4*(uint64_t)nquarts_linear_size*((uint64_t)nquarts_linear_size-1)-7)/2.0 - 0.5);
    uint64_t quarts_2 = gi + quarts_1 + 1 - (uint64_t)nquarts_linear_size*((uint64_t)nquarts_linear_size-1)/2 + ((uint64_t)nquarts_linear_size-quarts_1)*(((uint64_t)nquarts_linear_size-quarts_1)-1)/2;
    
    if (gi >= n_comparison || quarts_1 == quarts_2) return;

    point before [KERNEL_SIZE_DUP];
    for (int i = 0; i < KERNEL_SIZE_DUP; i++) {before[i] = d_contours[quarts_1 * KERNEL_SIZE_DUP + i];}

    point after [KERNEL_SIZE_DUP];
    for (int i = 0; i < KERNEL_SIZE_DUP; i++) {after[i] = d_contours[quarts_2 * KERNEL_SIZE_DUP + i]; }

    for (int i = 0; i < KERNEL_SIZE_DUP; i++){
        int f = d_flags[quarts_2 * KERNEL_SIZE_DUP];
        for (int j = 0; j < KERNEL_SIZE_DUP; j++) {
            f &= !(before[i].x == after[j].x && before[i].y == after[j].y);
        }
        d_flags[quarts_2 * KERNEL_SIZE_DUP + i] = f;
    }

    // point point_before_1 = d_contours[quarts_1 * 4];
    // point point_before_2 = d_contours[quarts_1 * 4 + 1];
    // point point_before_3 = d_contours[quarts_1 * 4 + 2];
    // point point_before_4 = d_contours[quarts_1 * 4 + 3];

    // point point_after_1 = d_contours[quarts_2 * 4];
    // point point_after_2 = d_contours[quarts_2 * 4 + 1];
    // point point_after_3 = d_contours[quarts_2 * 4 + 2];
    // point point_after_4 = d_contours[quarts_2 * 4 + 3];

    // d_flags[quarts_2].x = d_flags[quarts_2].x && !(point_before_1.x == point_after_1.x && point_before_1.y == point_after_1.y) && !(point_before_2.x == point_after_1.x && point_before_2.y == point_after_1.y) && !(point_before_3.x == point_after_1.x && point_before_3.y == point_after_1.y) && !(point_before_4.x == point_after_1.x && point_before_4.y == point_after_1.y);
    // d_flags[quarts_2].y = d_flags[quarts_2].y && !(point_before_1.x == point_after_2.x && point_before_1.y == point_after_2.y) && !(point_before_2.x == point_after_2.x && point_before_2.y == point_after_2.y) && !(point_before_3.x == point_after_2.x && point_before_3.y == point_after_2.y) && !(point_before_4.x == point_after_2.x && point_before_4.y == point_after_2.y);
    // d_flags[quarts_2].z = d_flags[quarts_2].z && !(point_before_1.x == point_after_3.x && point_before_1.y == point_after_3.y) && !(point_before_2.x == point_after_3.x && point_before_2.y == point_after_3.y) && !(point_before_3.x == point_after_3.x && point_before_3.y == point_after_3.y) && !(point_before_4.x == point_after_3.x && point_before_4.y == point_after_3.y);
    // d_flags[quarts_2].w = d_flags[quarts_2].w && !(point_before_1.x == point_after_4.x && point_before_1.y == point_after_4.y) && !(point_before_2.x == point_after_4.x && point_before_2.y == point_after_4.y) && !(point_before_3.x == point_after_4.x && point_before_3.y == point_after_4.y) && !(point_before_4.x == point_after_4.x && point_before_4.y == point_after_4.y);
    
}

void filter_contour_duplicate_wrapper(point * d_contours, int * h_contours_sizes, Sizes * sizes, int ngroups = 1024, int lws = 256){
    int *d_flags;
    hipError_t err;

    #if PROFILE_DUP
    hipEvent_t start, stop;
    float time = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    #endif
    
    err = hipMalloc((void **)&d_flags, sizes->contours_linear_size * sizeof(int)); cuda_err_check(err, __FILE__, __LINE__);
    hipMemsetD32((hipDeviceptr_t)d_flags, 1, sizes->contours_linear_size);

    uint64_t nquarts = round_div_up_64((uint64_t)sizes->contours_linear_size, KERNEL_SIZE_DUP);
    uint64_t nels = ((uint64_t)nquarts * ((uint64_t)nquarts - 1)) / 2;
    uint64_t lws_dup = 128;
    uint64_t gws = round_div_up_64(nels, lws_dup);

    #if PROFILE_DUP
    hipEventRecord(start);
    #endif

    compute_duplicates_flags<<<gws, lws_dup>>>(d_contours, d_flags, nquarts);

    #if PROFILE_DUP
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    int flag_accesses = 2 * nels * KERNEL_SIZE_DUP;
    uint64_t read_accesses = nels * 2 * KERNEL_SIZE_DUP;
    printf("Time for duplicate flags: %f ms\n", time);
    printf("GE/s: %f\n", (float)nels / time / 1e6);
    printf("GB/s: %f\n", (flag_accesses * sizeof(int) + read_accesses * sizeof(point)) / time / 1e6);
    #endif

    err = hipGetLastError(); cuda_err_check(err, __FILE__, __LINE__);
    err = hipDeviceSynchronize(); cuda_err_check(err, __FILE__, __LINE__);

    #if PRINT_DUP_FLAGS
    printf("Flags computed: ");
    print_array_dev(d_flags, sizes->contours_linear_size);
    printf("\n");
    #endif

    point * d_contours_out;

    err = hipMalloc((void **)&d_contours_out, sizes->contours_linear_size * sizeof(point)); cuda_err_check(err, __FILE__, __LINE__);

    #if PRINT_DUP_FLAGS
    printf("Before filter:  ");
    print_array_dev(d_contours_x, sizes->contours_linear_size);
    printf("\n");
    #endif

    filter_contour(d_contours, h_contours_sizes, d_contours_out, d_flags, sizes, ngroups, lws);

    #if PRINT_DUP_FLAGS
    printf("After filter:   ");
    print_array_dev(d_contours_x_out, sizes->contours_linear_size);
    printf("\n");
    #endif

    err = hipMemcpy(d_contours, d_contours_out, sizes->contours_linear_size * sizeof(point), hipMemcpyDeviceToDevice); cuda_err_check(err, __FILE__, __LINE__);

    err = hipFree(d_contours_out); cuda_err_check(err, __FILE__, __LINE__);
    err = hipFree(d_flags); cuda_err_check(err, __FILE__, __LINE__);

    #if PROFILE_DUP
    hipEventDestroy(start);
    hipEventDestroy(stop);
    #endif

    return;
}

#if 0

void test_duplicate (){
    // generate array of 100000000 random point numbers from 0 to 10
    
    // choose random number from 100 to 10000
    cout << "running test_duplicate\n";

    int n = rand() % 10000 + 100;

    Sizes * sizes = new Sizes;
    sizes->contours_linear_size = 70000;
    sizes->number_of_contours = n;

    point *points = new point[sizes->contours_linear_size];
    for (int i = 0; i < sizes->contours_linear_size; i++){
        points[i].x = rand() % 10;
        points[i].y = rand() % 10;
    }

    point * d_points;

    hipMalloc((void **)&d_points, sizes->contours_linear_size * sizeof(point));
    hipMemcpy(d_points, points, sizes->contours_linear_size * sizeof(point), hipMemcpyHostToDevice);

    cout << "transferred points to device\n";

    int * h_contours_sizes = new int[sizes->number_of_contours];

    for (int i = 0; i < sizes->number_of_contours - 1; i++) {
        h_contours_sizes[i] = 1;
    }
    h_contours_sizes[sizes->number_of_contours - 1] = sizes->contours_linear_size - sizes->number_of_contours;

    cout << "Calling filter_contour_duplicate_wrapper\n";

    filter_contour_duplicate_wrapper(d_points, h_contours_sizes, sizes, 256, 256);

    print_array_dev(d_points, sizes->contours_linear_size);

    hipFree(d_points);
    delete[] points;
    delete[] h_contours_sizes;
    delete sizes;
}

void test_duplicate () {
    int h_contours_x [] = {4,5,1,2,4,3,1};
    int h_contours_y [] = {4,5,1,2,4,3,1};

    int h_contours_sizes [] = {3, 2, 2};

    int * d_contours_x, * d_contours_y;
    
    hipError_t err;

    err = hipMalloc((void **)&d_contours_x, 7 * sizeof(int)); cuda_err_check(err, __FILE__, __LINE__);
    err = hipMalloc((void **)&d_contours_y, 7 * sizeof(int)); cuda_err_check(err, __FILE__, __LINE__);

    err = hipMemcpy(d_contours_x, h_contours_x, 7 * sizeof(int), hipMemcpyHostToDevice); cuda_err_check(err, __FILE__, __LINE__);
    err = hipMemcpy(d_contours_y, h_contours_y, 7 * sizeof(int), hipMemcpyHostToDevice); cuda_err_check(err, __FILE__, __LINE__);

    Sizes sizes;

    sizes.contours_linear_size = 7;
    sizes.number_of_contours = 3;

    printf("Before compute: ");
    print_array_dev(d_contours_x, sizes.contours_linear_size);
    printf("\n");

    // filter_contour_duplicate_wrapper(d_contours_x, d_contours_y, h_contours_sizes, &sizes);

    printf("After: ");
    print_array_dev(d_contours_x, sizes.contours_linear_size);
    printf("\n");
}
#endif

#endif