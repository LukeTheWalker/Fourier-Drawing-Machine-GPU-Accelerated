#include "hip/hip_runtime.h"
#ifndef FILTER_CONTOUR_BY_HAND_WRAPPER_H
#define FILTER_CONTOUR_BY_HAND_WRAPPER_H

#define PRINT_FLAGS 0
#define PROFILING_HAND 0

#include <hip/hip_runtime.h>

#include <vector>
#include <unordered_set>

#include "opencv2/core.hpp"
#include "opencv2/imgproc.hpp"
#include "contour.hpp"

#include "utils.cuh"
#include "filter_contour.cu"
#include "streamCompaction.cu"

using namespace std;

struct check_array_membership {
    __device__ int4 operator()(int gi, point * dat_arr, point * arr, int array_size) {
        int4 res = {0, 0, 0, 0};
        point dat_1 = dat_arr[gi * 4];
        point dat_2 = dat_arr[gi * 4 + 1];
        point dat_3 = dat_arr[gi * 4 + 2];
        point dat_4 = dat_arr[gi * 4 + 3];

        for (int i = 0; i < array_size; i++){
            point p = arr[i];
            res.x = res.x | (p.x == dat_1.x && p.y == dat_1.y);
            res.y = res.y | (p.x == dat_2.x && p.y == dat_2.y);
            res.z = res.z | (p.x == dat_3.x && p.y == dat_3.y);
            res.w = res.w | (p.x == dat_4.x && p.y == dat_4.y);
        }
        return {!res.x, !res.y, !res.z, !res.w};
    }
};

void load_contours_to_device (point * d_contours, vector<vector<Point>> &contours, Sizes * sizes) {
    point *h_contours;
    hipError_t err;

    h_contours = (point *)malloc(sizes->contours_linear_size * sizeof(point));
    

    int idx = 0;
    for (int i = 0; i < sizes->number_of_contours; i++){
        for (int j = 0; j < contours[i].size(); j++){
            h_contours[idx] = {contours[i][j].x, contours[i][j].y};

            idx++;
        }
    }
    
    err = hipMemcpy(d_contours, h_contours, sizes->contours_linear_size * sizeof(point), hipMemcpyHostToDevice); cuda_err_check(err, __FILE__, __LINE__);
    
    free(h_contours);
}

void filter_contour_by_hand_wrapper(point * d_contours_out, int * h_contours_sizes_out, vector<vector<Point>> &contours, unordered_set<Point, HashFunction> &_excluded_points, Sizes * sizes, int ngroups = 1024, int lws = 256){
    point *d_excluded_points, *d_contours;
    point *h_excluded_points;
    int *d_flags;
    int excluded_points_size = _excluded_points.size();
    hipError_t err;

    // allocate memory on the host for all the points
    h_excluded_points= (point *)malloc(excluded_points_size * sizeof(point));
    
    err = hipMalloc((void **)&d_contours, sizes->contours_linear_size * sizeof(point)); cuda_err_check(err, __FILE__, __LINE__);

    load_contours_to_device(d_contours, contours, sizes);

    int idx = 0;
    for (auto it = _excluded_points.begin(); it != _excluded_points.end(); it++){
        h_excluded_points[idx] = {it->x, it->y};
        idx++;
    }

    err = hipMalloc((void **)&d_excluded_points, excluded_points_size * sizeof(point)); cuda_err_check(err, __FILE__, __LINE__);
    err = hipMalloc((void **)&d_flags, sizes->contours_linear_size * sizeof(int)); cuda_err_check(err, __FILE__, __LINE__);

    err = hipMemcpy(d_excluded_points, h_excluded_points, excluded_points_size * sizeof(point), hipMemcpyHostToDevice); cuda_err_check(err, __FILE__, __LINE__);

    int nquarts_flags = round_div_up(sizes->contours_linear_size, 4);

    #if PROFILING_HAND
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    #endif

    compute_flags<check_array_membership><<<round_div_up(nquarts_flags, 256), 256>>>(nquarts_flags, (int4*)d_flags, d_contours, d_excluded_points, excluded_points_size);
    
    #if PROFILING_HAND
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("compute_flags hand time: %f\n", milliseconds);
    printf("GE/s: %f\n", (float)sizes->contours_linear_size / milliseconds / 1e6);
    printf("GB/s: %f\n", ((float)sizes->contours_linear_size * sizeof(int) * 3 + (float)sizes->contours_linear_size * excluded_points_size * sizeof(int))/ milliseconds / 1e6);
    #endif

    err = hipGetLastError(); cuda_err_check(err, __FILE__, __LINE__);
    err = hipDeviceSynchronize(); cuda_err_check(err, __FILE__, __LINE__);

    #if PRINT_FLAGS
    printf("Flags computed: ");
    print_array_dev(d_flags, sizes->contours_linear_size);
    printf("\n");
    #endif

    filter_contour(d_contours, h_contours_sizes_out, d_contours_out, d_flags, sizes, ngroups, lws);

    free(h_excluded_points);

    hipFree(d_contours);
    hipFree(d_excluded_points);
    hipFree(d_flags);

    #if PROFILING_HAND
    hipEventDestroy(start);
    hipEventDestroy(stop);
    #endif

    return;
}

int test() {
    vector<vector<Point>> contours;
    unordered_set<Point, HashFunction> excluded_points;

    ifstream in("debug.txt");
    int n;
    in >> n;
    vector<int> contour_sizes;

    for (int i = 0; i < n; i++){
        int m;
        in >> m;
        contour_sizes.push_back(m);
    }

    for (int i = 0; i < n; i++){
        vector<Point> contour;
        for (int j = 0; j < contour_sizes[i]; j++){
            int x, y;
            in >> x >> y;
            contour.push_back(Point(x, y));
        }
        contours.push_back(contour);
    }

    int m;
    in >> m;
    for (int i = 0; i < m; i++){
        int x, y;
        in >> x >> y;
        excluded_points.insert(Point(x, y));
    }

    cerr << "MISSING IMPORTANT DATA" << endl;

    int *d_contours_x, *d_contours_y;
    int *h_contours_sizes;
    hipError_t err;
    Sizes * sizes;

    sizes = (Sizes*)malloc(sizeof(Sizes));
    sizes->number_of_contours = contours.size();
    sizes->contours_linear_size = 0;

    for (int i = 0; i < contours.size(); i++) sizes->contours_linear_size += contours[i].size();

    err = hipMalloc((void **)&d_contours_x, sizes->contours_linear_size * sizeof(int)); cuda_err_check(err, __FILE__, __LINE__);
    err = hipMalloc((void **)&d_contours_y, sizes->contours_linear_size * sizeof(int)); cuda_err_check(err, __FILE__, __LINE__);

    h_contours_sizes = (int*)malloc(sizes->number_of_contours * sizeof(int));

    for (int i = 0; i < sizes->number_of_contours; i++) h_contours_sizes[i] = contours[i].size();


    // filter_contour_by_hand_wrapper(d_contours, d_contours_y, h_contours_sizes, contours, excluded_points, sizes);

    for (int i = 0; i < contours.size(); i++){
        for (int j = 0; j < contours[i].size(); j++){
            printf("(%d, %d) ", contours[i][j].x, contours[i][j].y);
        }
        printf("\n");
    }
    return 0;
}
#endif