#include "hip/hip_runtime.h"
#ifndef FILTER_CONTOUR_H
#define FILTER_CONTOUR_H

#define PRINT_CONTOUR 0
#define PROFILING_CONTOUR 0

#include <hip/hip_runtime.h>

#include <vector>
#include <unordered_set>

#include "opencv2/core.hpp"
#include "opencv2/imgproc.hpp"
#include "contour.hpp"

#include "utils.cuh"
#include "streamCompaction.cu"

__global__ void move_contours (int *d_contours_x, int *d_contours_y, int *dest_x, int *dest_y, int *d_flags, int *d_positions, int nels){
    int gi = threadIdx.x + blockIdx.x * blockDim.x;
    if (gi >= nels) return;
    if (!d_flags[gi]) return;
    if (gi == 0) { dest_x[0] = d_contours_x[0]; dest_y[0] = d_contours_y[0]; return; }
    
    int pos = d_positions[gi - 1];

    dest_x[pos] = d_contours_x[gi];
    dest_y[pos] = d_contours_y[gi];
}

void filter_contour (int * d_contours_x, int * d_contours_y, int * h_contours_sizes,  int * d_contours_x_out, int * d_contours_y_out, int * d_flags, Sizes * sizes, int ngroups, int lws){
    hipError_t err;
    int *d_positions, *d_tails;

    int ntails = ngroups > 1 ? round_mul_up(ngroups, 4) : ngroups;
    err = hipMalloc((void **)&d_positions, sizes->contours_linear_size * sizeof(int)); cuda_err_check(err, __FILE__, __LINE__);
    err = hipMalloc((void **)&d_tails, ngroups * sizeof(int)); cuda_err_check(err, __FILE__, __LINE__);

    #if PROFILING_CONTOUR
    hipEvent_t start, stop;
    float time;
    err = hipEventCreate(&start); cuda_err_check(err, __FILE__, __LINE__);
    err = hipEventCreate(&stop); cuda_err_check(err, __FILE__, __LINE__);
    err = hipEventRecord(start, 0); cuda_err_check(err, __FILE__, __LINE__);
    #endif

    scan_sliding_window<<<ngroups, lws, lws*sizeof(int)>>>((int4*)d_flags, (int4*)d_positions, d_tails, round_div_up(sizes->contours_linear_size, 4), 32);
    #if PROFILING_CONTOUR
    err = hipEventRecord(stop, 0); cuda_err_check(err, __FILE__, __LINE__);
    err = hipEventSynchronize(stop); cuda_err_check(err, __FILE__, __LINE__);
    err = hipEventElapsedTime(&time, start, stop); cuda_err_check(err, __FILE__, __LINE__);
    printf("scan_sliding_window[partial] time: %f\n", time);
    printf("GE/s = %f\n", (float)sizes->contours_linear_size / time / 1e6);
    printf("GB/s = %f\n", (2 * (float)sizes->contours_linear_size * sizeof(int) + (ngroups > 1 ? (float)ngroups * sizeof(int) : 0)) / time / 1.e6);
    #endif
    
    err = hipGetLastError(); cuda_err_check(err, __FILE__, __LINE__);
    err = hipDeviceSynchronize(); cuda_err_check(err, __FILE__, __LINE__);

    #if PRINT_CONTOUR
    printf("Positions computed: ");
    print_array_dev(d_positions, sizes->contours_linear_size);
    printf("\n");
    #endif


    
    if (ngroups > 1){
        #if PROFILING_CONTOUR
        err = hipEventRecord(start, 0); cuda_err_check(err, __FILE__, __LINE__);
        #endif

        scan_sliding_window<<<1, lws, lws*sizeof(int)>>>((int4*)d_tails, (int4*)d_tails, NULL, round_div_up(ntails, 4), 32);
        #if PROFILING_CONTOUR
        err = hipEventRecord(stop, 0); cuda_err_check(err, __FILE__, __LINE__);
        err = hipEventSynchronize(stop); cuda_err_check(err, __FILE__, __LINE__);
        err = hipEventElapsedTime(&time, start, stop); cuda_err_check(err, __FILE__, __LINE__);
        printf("scan_sliding_window time: %f\n", time);
        printf("GE/s = %f\n", (float)ntails / time / 1e6);
        printf("GB/s = %f\n", (float)ntails * sizeof(int) / time / 1.e6);
        #endif

        err = hipGetLastError(); cuda_err_check(err, __FILE__, __LINE__);
    }

    #if PRINT_CONTOUR
    printf("Tails computed: ");
    print_array_dev(d_tails, ngroups);
    printf("\n");
    #endif

    if (ngroups > 1){
        #if PROFILING_CONTOUR
        err = hipEventRecord(start, 0); cuda_err_check(err, __FILE__, __LINE__);
        #endif

        scan_fixup<<<ngroups, lws>>>((int4*)d_positions, d_tails, round_div_up(sizes->contours_linear_size, 4), 32);

        #if PROFILING_CONTOUR
        err = hipEventRecord(stop, 0); cuda_err_check(err, __FILE__, __LINE__);
        err = hipEventSynchronize(stop); cuda_err_check(err, __FILE__, __LINE__);
        err = hipEventElapsedTime(&time, start, stop); cuda_err_check(err, __FILE__, __LINE__);
        printf("scan_fixup time: %f\n", time);
        printf("GE/s = %f\n", (float)sizes->contours_linear_size / time / 1e6);
        printf("GB/s = %f\n", (2*(float)(sizes->contours_linear_size - lws) + ngroups) * sizeof(int) / time / 1.e6);
        #endif

        err = hipGetLastError(); cuda_err_check(err, __FILE__, __LINE__);
    }

    #if PRINT_CONTOUR
    printf("Positions computed: ");
    print_array_dev(d_positions, sizes->contours_linear_size);
    printf("\n");
    #endif

    #if PROFILING_CONTOUR
    err = hipEventRecord(start, 0); cuda_err_check(err, __FILE__, __LINE__);
    #endif

    move_contours<<<round_div_up(sizes->contours_linear_size, lws), lws>>>(d_contours_x, d_contours_y, d_contours_x_out, d_contours_y_out, d_flags, d_positions, sizes->contours_linear_size);

    #if PROFILING_CONTOUR
    err = hipEventRecord(stop, 0); cuda_err_check(err, __FILE__, __LINE__);
    err = hipEventSynchronize(stop); cuda_err_check(err, __FILE__, __LINE__);
    err = hipEventElapsedTime(&time, start, stop); cuda_err_check(err, __FILE__, __LINE__);
    printf("move_contours time: %f\n", time);
    printf("GE/s = %f\n", (float)sizes->contours_linear_size / time / 1e6);
    printf("GB/s = %f\n", 6 * (float)sizes->contours_linear_size * sizeof(int) / time / 1.e6);
    #endif

    err = hipGetLastError(); cuda_err_check(err, __FILE__, __LINE__);
    err = hipDeviceSynchronize(); cuda_err_check(err, __FILE__, __LINE__);

    #if PRINT_CONTOUR
    printf("Contour x computed: ");
    print_array_dev(d_contours_x_out, sizes->contours_linear_size);
    printf("\n");
    #endif

    int *h_positions;

    err = hipHostMalloc(&h_positions, sizes->contours_linear_size * sizeof(int)); cuda_err_check(err, __FILE__, __LINE__);
    err = hipMemcpy(h_positions, d_positions, sizes->contours_linear_size * sizeof(int), hipMemcpyDeviceToHost); cuda_err_check(err, __FILE__, __LINE__);

    uint32_t cnt = 0;
    uint32_t cnt_n_contours = 0;
    for (int i = 0; i < sizes->number_of_contours; i++){
        cnt += h_contours_sizes[i];
        int tmp_contour_size = h_positions[cnt-1] - (i == 0 ? 0 : h_positions[cnt - h_contours_sizes[i] - 1] );
        
        if (tmp_contour_size > 0){
            h_contours_sizes[cnt_n_contours] = tmp_contour_size;
            cnt_n_contours++;
        }
    }

    sizes->contours_linear_size = h_positions[sizes->contours_linear_size - 1];
    sizes->number_of_contours = cnt_n_contours;

    err = hipFree(d_positions); cuda_err_check(err, __FILE__, __LINE__);
    err = hipFree(d_tails); cuda_err_check(err, __FILE__, __LINE__);

    err = hipHostFree(h_positions); cuda_err_check(err, __FILE__, __LINE__);
}

#endif